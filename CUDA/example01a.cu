#include "hip/hip_runtime.h"
// =================================================================
//
// File: example01.cpp
// Author: Pedro Perez
// Description: This file implements the addition of two vectors. 
//				The time this implementation takes will be used as 
//				the basis to calculate the improvement obtained with 
//				parallel technologies.
//
// Copyright (c) 2024 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <iostream>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>
#include "utils.h"

using namespace std;
using namespace std::chrono;

const int SIZE = 1000000; // 1e6

__global__ void add_vector(int *c, int *a, int *b) {
    //Threads
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(int argc, char* argv[]) {
    int *a, *b, *c;
    int *device_a, *device_b, *device_c;

    // These variables are used to keep track of the execution time.
    high_resolution_clock::time_point start, end;
    double timeElapsed;

    //We create the arrays.
    a = new int [SIZE];
    b = new int [SIZE];
    c = new int [SIZE];

    hipMalloc((void**) &device_a, SIZE * sizeof(int));
    hipMalloc((void**) &device_b, SIZE * sizeof(int));
    hipMalloc((void**) &device_c, SIZE * sizeof(int));

    fill_array(a, SIZE);
    display_array("a:", a);
    fill_array(b, SIZE);
    display_array("b:", b);

    hipMemcpy(device_a, a, SIZE*sizeof(int),
        hipMemcpyHostToDevice);

    hipMemcpy(device_b, b, SIZE*sizeof(int),
        hipMemcpyHostToDevice);

    cout << "Starting...\n";
    timeElapsed = 0;
    for (int j = 0; j < N; j++) {
        start = high_resolution_clock::now();

        add_vector<<<1, SIZE>>>(device_c, device_a, device_b);

        end = high_resolution_clock::now();
        timeElapsed += 
            duration<double, std::milli>(end - start).count();
    }

    hipMemcpy(c, device_c, SIZE * sizeof(int),
        hipMemcpyDeviceToHost);

    display_array("c:", c);
    cout << "avg time = " << fixed << setprecision(3) 
         << (timeElapsed / N) <<  " ms\n";

    delete [] a;
    delete [] b;
    delete [] c;

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}